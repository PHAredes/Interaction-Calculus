#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hvmn.h"

// Maximum threads per block (hardware constraint)
#define HVMN_MAX_THREADS_PER_BLOCK 512

// Check if CUDA is available
extern "C" int hvmn_cuda_available() {
  int deviceCount = 0;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) {
    return 0; // CUDA error occurred
  }
  return deviceCount > 0;
}

// Device memory pointers and sizes
__device__ Term* d_heap;
__device__ Term* d_stack;
__device__ uint32_t d_heap_size_per_thread;
__device__ uint32_t d_stack_size_per_thread;
__device__ uint32_t* d_heap_pos_array;          // Per-thread heap positions
__device__ unsigned long long* d_interactions_array; // Per-thread interaction counters

// Device utility functions

// Create a term with substitution bit
__device__ inline Term d_hvmn_make_sub(Term term) {
  return term | TERM_SUB_MASK;
}

// Remove substitution bit from a term
__device__ inline Term d_hvmn_clear_sub(Term term) {
  return term & ~TERM_SUB_MASK;
}

// Create a term with specified tag, label, and value
__device__ inline Term d_hvmn_make_term(bool sub, TermTag tag, uint8_t lab, uint32_t val) {
  return MAKE_TERM(sub, tag, lab, val);
}

// Allocate n consecutive terms in the thread's local heap
__device__ inline uint32_t d_hvmn_alloc(uint32_t n, uint32_t* heap_pos) {
  if (*heap_pos + n > d_heap_size_per_thread) {
    // Handle overflow - return current position without updating
    // This prevents illegal memory access but may cause computation errors
    return *heap_pos;
  }
  
  uint32_t ptr = *heap_pos;
  *heap_pos += n;
  return ptr;
}

// Apply a lambda to an argument
__device__ inline Term d_hvmn_app_lam(Term app, Term lam, Term* heap, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t app_loc = TERM_VAL(app);
  uint32_t lam_loc = TERM_VAL(lam);
  Term arg = heap[app_loc + 1];
  Term bod = heap[lam_loc + 0];
  heap[lam_loc] = d_hvmn_make_sub(arg);
  return bod;
}

// Apply a superposition
__device__ inline Term d_hvmn_app_sup(Term app, Term sup, Term* heap, uint32_t* heap_pos, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t app_loc = TERM_VAL(app);
  uint32_t sup_loc = TERM_VAL(sup);
  uint8_t sup_lab = TERM_LAB(sup);
  Term arg = heap[app_loc + 1];
  Term lft = heap[sup_loc + 0];
  Term rgt = heap[sup_loc + 1];

  uint32_t col_loc = d_hvmn_alloc(1, heap_pos);
  uint32_t app1_loc = d_hvmn_alloc(2, heap_pos);
  heap[col_loc] = arg;

  Term x0 = d_hvmn_make_term(false, CO0, sup_lab, col_loc);
  Term x1 = d_hvmn_make_term(false, CO1, sup_lab, col_loc);

  heap[sup_loc + 1] = x0; // Reuse sup_loc as app0_loc
  heap[app1_loc + 0] = rgt;
  heap[app1_loc + 1] = x1;

  heap[app_loc + 0] = d_hvmn_make_term(false, APP, 0, sup_loc);
  heap[app_loc + 1] = d_hvmn_make_term(false, APP, 0, app1_loc);
  return d_hvmn_make_term(false, SUP, sup_lab, app_loc);
}

// Collapse a lambda
__device__ inline Term d_hvmn_col_lam(Term col, Term lam, Term* heap, uint32_t* heap_pos, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t col_loc = TERM_VAL(col);
  uint32_t lam_loc = TERM_VAL(lam);
  uint8_t col_lab = TERM_LAB(col);
  bool is_co0 = (TERM_TAG(col) == CO0);
  Term bod = heap[lam_loc + 0];

  uint32_t alloc_start = d_hvmn_alloc(5, heap_pos);
  uint32_t lam0_loc = alloc_start;
  uint32_t lam1_loc = alloc_start + 1;
  uint32_t sup_loc = alloc_start + 2;
  uint32_t col_new_loc = alloc_start + 4;

  heap[sup_loc + 0] = d_hvmn_make_term(false, VAR, 0, lam0_loc);
  heap[sup_loc + 1] = d_hvmn_make_term(false, VAR, 0, lam1_loc);
  heap[lam_loc] = d_hvmn_make_sub(d_hvmn_make_term(false, SUP, col_lab, sup_loc));
  heap[col_new_loc] = bod;
  heap[lam0_loc] = d_hvmn_make_term(false, CO0, col_lab, col_new_loc);
  heap[lam1_loc] = d_hvmn_make_term(false, CO1, col_lab, col_new_loc);

  if (is_co0) {
    heap[col_loc] = d_hvmn_make_sub(d_hvmn_make_term(false, LAM, 0, lam1_loc));
    return d_hvmn_make_term(false, LAM, 0, lam0_loc);
  } else {
    heap[col_loc] = d_hvmn_make_sub(d_hvmn_make_term(false, LAM, 0, lam0_loc));
    return d_hvmn_make_term(false, LAM, 0, lam1_loc);
  }
}

// Collapse a superposition
__device__ inline Term d_hvmn_col_sup(Term col, Term sup, Term* heap, uint32_t* heap_pos, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t col_loc = TERM_VAL(col);
  uint32_t sup_loc = TERM_VAL(sup);
  uint8_t col_lab = TERM_LAB(col);
  uint8_t sup_lab = TERM_LAB(sup);
  bool is_co0 = (TERM_TAG(col) == CO0);
  Term lft = heap[sup_loc + 0];
  Term rgt = heap[sup_loc + 1];

  if (col_lab == sup_lab) {
    if (is_co0) {
      heap[col_loc] = d_hvmn_make_sub(rgt);
      return lft;
    } else {
      heap[col_loc] = d_hvmn_make_sub(lft);
      return rgt;
    }
  } else {
    uint32_t sup_start = d_hvmn_alloc(4, heap_pos);
    uint32_t sup0_loc = sup_start;
    uint32_t sup1_loc = sup_start + 2;

    heap[sup0_loc + 0] = d_hvmn_make_term(false, CO0, col_lab, sup_loc + 0);
    heap[sup0_loc + 1] = d_hvmn_make_term(false, CO0, col_lab, sup_loc + 1);
    heap[sup1_loc + 0] = d_hvmn_make_term(false, CO1, col_lab, sup_loc + 0);
    heap[sup1_loc + 1] = d_hvmn_make_term(false, CO1, col_lab, sup_loc + 1);
    heap[sup_loc + 0] = lft;
    heap[sup_loc + 1] = rgt;

    if (is_co0) {
      heap[col_loc] = d_hvmn_make_sub(d_hvmn_make_term(false, SUP, sup_lab, sup1_loc));
      return d_hvmn_make_term(false, SUP, sup_lab, sup0_loc);
    } else {
      heap[col_loc] = d_hvmn_make_sub(d_hvmn_make_term(false, SUP, sup_lab, sup0_loc));
      return d_hvmn_make_term(false, SUP, sup_lab, sup1_loc);
    }
  }
}

// Reduce a term to WHNF (Weak Head Normal Form)
__device__ inline Term d_hvmn_whnf(Term term, Term* heap, Term* stack, uint32_t* heap_pos, uint32_t* stack_pos, unsigned long long* interactions) {
  uint32_t stop = *stack_pos;
  Term next = term;
  uint32_t local_stack_pos = stop;

  while (true) {
    TermTag tag = TERM_TAG(next);
    switch (tag) {
      case VAR: {
        uint32_t var_loc = TERM_VAL(next);
        Term subst = heap[var_loc];
        if (TERM_SUB(subst)) {
          next = d_hvmn_clear_sub(subst);
          continue;
        }
        break;
      }
      case CO0:
      case CO1: {
        uint32_t col_loc = TERM_VAL(next);
        Term val = heap[col_loc];
        if (TERM_SUB(val)) {
          next = d_hvmn_clear_sub(val);
          continue;
        } else {
          stack[local_stack_pos++] = next;
          next = val;
          continue;
        }
      }
      case APP: {
        uint32_t app_loc = TERM_VAL(next);
        stack[local_stack_pos++] = next;
        next = heap[app_loc];
        continue;
      }
      default: {
        if (local_stack_pos == stop) {
          *stack_pos = local_stack_pos;
          return next;
        }
        Term prev = stack[--local_stack_pos];
        TermTag ptag = TERM_TAG(prev);
        if (ptag == APP && tag == LAM) {
          next = d_hvmn_app_lam(prev, next, heap, interactions);
          continue;
        } else if (ptag == APP && tag == SUP) {
          next = d_hvmn_app_sup(prev, next, heap, heap_pos, interactions);
          continue;
        } else if ((ptag == CO0 || ptag == CO1) && tag == LAM) {
          next = d_hvmn_col_lam(prev, next, heap, heap_pos, interactions);
          continue;
        } else if ((ptag == CO0 || ptag == CO1) && tag == SUP) {
          next = d_hvmn_col_sup(prev, next, heap, heap_pos, interactions);
          continue;
        }
        stack[local_stack_pos++] = prev;
        break;
      }
    }
    if (local_stack_pos == stop) {
      *stack_pos = local_stack_pos;
      return next;
    }
    while (local_stack_pos > stop) {
      Term host = stack[--local_stack_pos];
      TermTag htag = TERM_TAG(host);
      uint32_t hloc = TERM_VAL(host);
      if (htag == APP || htag == CO0 || htag == CO1) {
        heap[hloc] = next;
      }
      next = host;
    }
    *stack_pos = local_stack_pos;
    return next;
  }
}

// Reduce a term to normal form
__device__ inline Term d_hvmn_normal(Term term, Term* heap, Term* stack, uint32_t* heap_pos, uint32_t* stack_pos, unsigned long long* interactions) {
  *stack_pos = 0;
  uint32_t local_stack_pos = 0;
  uint32_t root_loc = d_hvmn_alloc(1, heap_pos);
  heap[root_loc] = term;
  stack[local_stack_pos++] = d_hvmn_make_term(false, (TermTag)0, 0, root_loc);

  while (local_stack_pos > 0) {
    uint32_t loc = TERM_VAL(stack[--local_stack_pos]);
    Term current = heap[loc];
    current = d_hvmn_whnf(current, heap, stack, heap_pos, &local_stack_pos, interactions);
    heap[loc] = current;
    TermTag tag = TERM_TAG(current);
    uint32_t val = TERM_VAL(current);
    if (tag == LAM) {
      stack[local_stack_pos++] = d_hvmn_make_term(false, (TermTag)0, 0, val);
    } else if (tag == APP || tag == SUP) {
      stack[local_stack_pos++] = d_hvmn_make_term(false, (TermTag)0, 0, val);
      stack[local_stack_pos++] = d_hvmn_make_term(false, (TermTag)0, 0, val + 1);
    }
  }
  *stack_pos = local_stack_pos;
  return heap[root_loc];
}

// CUDA kernel for normalization
__global__ void normalizeKernel(int N, uint32_t initial_size) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= N) return;

  // Local heap and stack for this thread
  Term* local_heap = d_heap + (tid * d_heap_size_per_thread);
  Term* local_stack = d_stack + (tid * d_stack_size_per_thread);
  uint32_t* local_heap_pos = &d_heap_pos_array[tid];
  uint32_t local_stack_pos = 0;
  unsigned long long* local_interactions = &d_interactions_array[tid];

  // Copy initial term to local heap
  for (uint32_t i = 0; i < initial_size; i++) {
    Term original = d_heap[i];
    bool sub = TERM_SUB(original);
    TermTag tag = TERM_TAG(original);
    uint8_t lab = TERM_LAB(original);
    uint32_t val = TERM_VAL(original);
    local_heap[i] = d_hvmn_make_term(sub, tag, lab, val);
  }
  *local_heap_pos = initial_size;

  // Normalize the term
  Term term = local_heap[0];
  term = d_hvmn_normal(term, local_heap, local_stack, local_heap_pos, &local_stack_pos, local_interactions);
  local_heap[0] = term;
}

// Host function to normalize a term on the GPU
extern "C" Term hvmn_normal_cuda(HVMN* hvmn, Term term, int thread_count) {
  const size_t TOTAL_BUFFER_SIZE = 22ULL * 1024ULL * 1024ULL * 1024ULL; // 22 GB
  const size_t HEAP_PORTION = 18ULL * 1024ULL * 1024ULL * 1024ULL; // 18 GB for heaps
  const size_t STACK_PORTION = 4ULL * 1024ULL * 1024ULL * 1024ULL; // 4 GB for stacks
  
  int N = (thread_count > 0) ? thread_count : 1;
  
  const size_t HEAP_SIZE_PER_THREAD_BYTES = HEAP_PORTION / N;
  const size_t STACK_SIZE_PER_THREAD_BYTES = STACK_PORTION / N;
  
  const uint32_t heap_size_per_thread = HEAP_SIZE_PER_THREAD_BYTES / sizeof(Term);
  const uint32_t stack_size_per_thread = STACK_SIZE_PER_THREAD_BYTES / sizeof(Term);

  // Display memory allocation info for debug purposes
  printf("Memory Config: Total: %.2f GB (Heap: %.2f GB, Stack: %.2f GB)\n", 
         TOTAL_BUFFER_SIZE / (1024.0 * 1024.0 * 1024.0),
         HEAP_PORTION / (1024.0 * 1024.0 * 1024.0),
         STACK_PORTION / (1024.0 * 1024.0 * 1024.0));
  printf("Per Thread: Heap: %.2f MB, Stack: %.2f MB\n",
         HEAP_SIZE_PER_THREAD_BYTES / (1024.0 * 1024.0),
         STACK_SIZE_PER_THREAD_BYTES / (1024.0 * 1024.0));

  // Device memory allocation
  Term *d_heap_ptr, *d_stack_ptr;
  uint32_t* d_heap_pos_array_ptr;
  unsigned long long* d_interactions_array_ptr;
  hipError_t err;

  err = hipMalloc(&d_heap_ptr, HEAP_PORTION);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap): %s\n", hipGetErrorString(err));
    return term;
  }

  err = hipMalloc(&d_stack_ptr, STACK_PORTION);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (stack): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    return term;
  }

  err = hipMalloc(&d_heap_pos_array_ptr, N * sizeof(uint32_t));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap_pos): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    return term;
  }

  err = hipMalloc(&d_interactions_array_ptr, N * sizeof(unsigned long long));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (interactions): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    return term;
  }

  // Copy initial heap to device (temporary buffer for kernel to distribute)
  err = hipMemcpy(d_heap_ptr, hvmn->heap, hvmn->heap_pos * sizeof(Term), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (copy to device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    return term;
  }

  // Initialize arrays
  uint32_t* h_heap_pos_array = (uint32_t*)malloc(N * sizeof(uint32_t));
  unsigned long long* h_interactions_array = (unsigned long long*)malloc(N * sizeof(unsigned long long));
  for (int i = 0; i < N; i++) {
    h_heap_pos_array[i] = 0;
    h_interactions_array[i] = 0;
  }
  hipMemcpy(d_heap_pos_array_ptr, h_heap_pos_array, N * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_interactions_array_ptr, h_interactions_array, N * sizeof(unsigned long long), hipMemcpyHostToDevice);

  // Set device symbols
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap), &d_heap_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack), &d_stack_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_size_per_thread), &heap_size_per_thread, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack_size_per_thread), &stack_size_per_thread, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_pos_array), &d_heap_pos_array_ptr, sizeof(uint32_t*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_interactions_array), &d_interactions_array_ptr, sizeof(unsigned long long*));

  // Launch kernel
  int blocks = (N + HVMN_MAX_THREADS_PER_BLOCK - 1) / HVMN_MAX_THREADS_PER_BLOCK;
  int threads_per_block = (N < HVMN_MAX_THREADS_PER_BLOCK) ? N : HVMN_MAX_THREADS_PER_BLOCK;
  printf("CUDA Configuration: %d thread(s) total, %d block(s), %d thread(s) per block\n", N, blocks, threads_per_block);
  normalizeKernel<<<blocks, threads_per_block>>>(N, hvmn->heap_pos);
  hipDeviceSynchronize();

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Kernel Error: %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    free(h_heap_pos_array);
    free(h_interactions_array);
    return term;
  }

  // Copy results back
  hipMemcpy(h_heap_pos_array, d_heap_pos_array_ptr, N * sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(h_interactions_array, d_interactions_array_ptr, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);

  uint64_t total_interactions = 0;
  for (int i = 0; i < N; i++) {
    total_interactions += h_interactions_array[i];
  }
  uint32_t heap_pos_thread0 = h_heap_pos_array[0];

  // Only copy back the result from the first thread
  err = hipMemcpy(hvmn->heap, d_heap_ptr, heap_pos_thread0 * sizeof(Term), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (copy from device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    free(h_heap_pos_array);
    free(h_interactions_array);
    return term;
  }

  hvmn->heap_pos = heap_pos_thread0;
  hvmn->interactions = total_interactions;

  hipFree(d_heap_ptr);
  hipFree(d_stack_ptr);
  hipFree(d_heap_pos_array_ptr);
  hipFree(d_interactions_array_ptr);
  free(h_heap_pos_array);
  free(h_interactions_array);

  return hvmn->heap[0];
}

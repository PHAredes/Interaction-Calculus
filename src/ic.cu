#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "ic.h"

// Check if CUDA is available
extern "C" int ic_cuda_available() {
  int deviceCount = 0;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  
  if (error != hipSuccess) {
    return 0; // CUDA error occurred
  }
  
  return deviceCount > 0;
}

// Device memory for heap and stack
__device__ Term* d_heap;
__device__ Term* d_stack;
__device__ uint32_t d_heap_size;
__device__ uint32_t d_stack_size;
__device__ uint32_t d_heap_pos;
__device__ uint32_t d_stack_pos;
__device__ uint64_t d_interactions;

// Device implementations of IC functions

// Create a term with substitution bit
__device__ inline Term d_ic_make_sub(Term term) {
  return term | TERM_SUB_MASK;
}

// Remove substitution bit from a term
__device__ inline Term d_ic_clear_sub(Term term) {
  return term & ~TERM_SUB_MASK;
}

// Create a term with specified tag, label, and value
__device__ inline Term d_ic_make_term(TermTag tag, uint8_t lab, uint32_t val) {
  return MAKE_TERM(false, tag, lab, val);
}

// Allocate n consecutive terms in memory
__device__ inline uint32_t d_ic_alloc(uint32_t n) {
  uint32_t ptr = d_heap_pos;
  d_heap_pos += n;
  
  // Check if we've run out of memory
  if (d_heap_pos >= d_heap_size) {
    // In a real implementation, we'd need error handling here
    // Since we can't easily abort a kernel, we'll just wrap around
    // This is just a safeguard; the host should ensure enough memory
    d_heap_pos = d_heap_size - 1;
  }
  
  return ptr;
}

// Apply a lambda to an argument
__device__ inline Term d_ic_app_lam(Term app, Term lam) {
  d_interactions++;
  
  uint32_t app_loc = TERM_VAL(app);
  uint32_t lam_loc = TERM_VAL(lam);
  
  Term arg = d_heap[app_loc + 1];
  Term bod = d_heap[lam_loc + 0];

  // Create substitution for the lambda variable
  d_heap[lam_loc] = d_ic_make_sub(arg);

  return bod;
}

// Apply a superposition
__device__ inline Term d_ic_app_sup(Term app, Term sup) {
  d_interactions++;
  
  uint32_t app_loc = TERM_VAL(app);
  uint32_t sup_loc = TERM_VAL(sup);
  uint8_t sup_lab = TERM_LAB(sup);

  Term arg = d_heap[app_loc + 1];
  Term lft = d_heap[sup_loc + 0];
  Term rgt = d_heap[sup_loc + 1];

  // Allocate only what's necessary
  uint32_t col_loc = d_ic_alloc(1);
  uint32_t app1_loc = d_ic_alloc(2);
  
  // Store the arg in the collapser location
  d_heap[col_loc] = arg;

  // Create CO0 and CO1 terms
  Term x0 = d_ic_make_term(CO0, sup_lab, col_loc);
  Term x1 = d_ic_make_term(CO1, sup_lab, col_loc);

  // Reuse sup_loc for app0
  d_heap[sup_loc + 1] = x0; // lft is already in heap[sup_loc + 0]

  // Set up app1
  d_heap[app1_loc + 0] = rgt;
  d_heap[app1_loc + 1] = x1;

  // Reuse app_loc for the result superposition
  d_heap[app_loc + 0] = d_ic_make_term(APP, 0, sup_loc);
  d_heap[app_loc + 1] = d_ic_make_term(APP, 0, app1_loc);

  return d_ic_make_term(SUP, sup_lab, app_loc);
}

// Collapse a lambda
__device__ inline Term d_ic_col_lam(Term col, Term lam) {
  d_interactions++;
  
  uint32_t col_loc = TERM_VAL(col);
  uint32_t lam_loc = TERM_VAL(lam);
  uint8_t col_lab = TERM_LAB(col);
  uint8_t is_co0 = (TERM_TAG(col) == CO0);

  Term bod = d_heap[lam_loc + 0];

  // Batch allocate memory for efficiency
  uint32_t alloc_start = d_ic_alloc(5);
  uint32_t lam0_loc = alloc_start;
  uint32_t lam1_loc = alloc_start + 1;
  uint32_t sup_loc = alloc_start + 2; // 2 locations
  uint32_t col_new_loc = alloc_start + 4;

  // Set up the superposition
  d_heap[sup_loc + 0] = d_ic_make_term(VAR, 0, lam0_loc);
  d_heap[sup_loc + 1] = d_ic_make_term(VAR, 0, lam1_loc);

  // Replace lambda's variable with the superposition
  d_heap[lam_loc] = d_ic_make_sub(d_ic_make_term(SUP, col_lab, sup_loc));

  // Set up the new collapser
  d_heap[col_new_loc] = bod;

  // Set up new lambda bodies
  d_heap[lam0_loc] = d_ic_make_term(CO0, col_lab, col_new_loc);
  d_heap[lam1_loc] = d_ic_make_term(CO1, col_lab, col_new_loc);

  // Create and return the appropriate lambda
  if (is_co0) {
    d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(LAM, 0, lam1_loc));
    return d_ic_make_term(LAM, 0, lam0_loc);
  } else {
    d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(LAM, 0, lam0_loc));
    return d_ic_make_term(LAM, 0, lam1_loc);
  }
}

// Collapse a superposition
__device__ inline Term d_ic_col_sup(Term col, Term sup) {
  d_interactions++;
  
  uint32_t col_loc = TERM_VAL(col);
  uint32_t sup_loc = TERM_VAL(sup);
  uint8_t col_lab = TERM_LAB(col);
  uint8_t sup_lab = TERM_LAB(sup);
  uint8_t is_co0 = (TERM_TAG(col) == CO0);

  Term lft = d_heap[sup_loc + 0];
  Term rgt = d_heap[sup_loc + 1];

  // Fast path for matching labels (common case)
  if (col_lab == sup_lab) {
    // Labels match: simple substitution
    if (is_co0) {
      d_heap[col_loc] = d_ic_make_sub(rgt);
      return lft;
    } else {
      d_heap[col_loc] = d_ic_make_sub(lft);
      return rgt;
    }
  } else {
    // Labels don't match: create nested collapsers
    uint32_t sup_start = d_ic_alloc(4); // 2 sups with 2 terms each
    uint32_t sup0_loc = sup_start;
    uint32_t sup1_loc = sup_start + 2;

    // Use existing locations as collapser locations
    uint32_t col_lft_loc = sup_loc + 0;
    uint32_t col_rgt_loc = sup_loc + 1;
    
    // Set up the first superposition (for CO0)
    d_heap[sup0_loc + 0] = d_ic_make_term(CO0, col_lab, col_lft_loc);
    d_heap[sup0_loc + 1] = d_ic_make_term(CO0, col_lab, col_rgt_loc);
    
    // Set up the second superposition (for CO1)
    d_heap[sup1_loc + 0] = d_ic_make_term(CO1, col_lab, col_lft_loc);
    d_heap[sup1_loc + 1] = d_ic_make_term(CO1, col_lab, col_rgt_loc);
    
    // Set up original collapsers to point to lft and rgt
    d_heap[col_lft_loc] = lft;
    d_heap[col_rgt_loc] = rgt;

    if (is_co0) {
      d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(SUP, sup_lab, sup1_loc));
      return d_ic_make_term(SUP, sup_lab, sup0_loc);
    } else {
      d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(SUP, sup_lab, sup0_loc));
      return d_ic_make_term(SUP, sup_lab, sup1_loc);
    }
  }
}

// Reduce a term to WHNF (Weak Head Normal Form)
__device__ inline Term d_ic_whnf(Term term) {
  uint32_t stop = d_stack_pos;
  Term next = term;
  uint32_t stack_pos = stop;

  while (1) {
    TermTag tag = TERM_TAG(next);

    switch (tag) {
      case VAR: {
        uint32_t var_loc = TERM_VAL(next);
        Term subst = d_heap[var_loc];
        if (TERM_SUB(subst)) {
          next = d_ic_clear_sub(subst);
          continue;
        }
        break; // No substitution, so it's in WHNF
      }

      case CO0:
      case CO1: {
        uint32_t col_loc = TERM_VAL(next);
        Term val = d_heap[col_loc];
        if (TERM_SUB(val)) {
          next = d_ic_clear_sub(val);
          continue;
        } else {
          d_stack[stack_pos++] = next;
          next = val;
          continue;
        }
      }

      case APP: {
        uint32_t app_loc = TERM_VAL(next);
        d_stack[stack_pos++] = next;
        next = d_heap[app_loc]; // Reduce the function part
        continue;
      }

      default: { // SUP, LAM
        if (stack_pos == stop) {
          d_stack_pos = stack_pos; // Update stack position before return
          return next; // Stack empty, term is in WHNF
        } else {
          Term prev = d_stack[--stack_pos];
          TermTag ptag = TERM_TAG(prev);
          
          // Handle interactions based on term types
          if (ptag == APP && tag == LAM) {
            next = d_ic_app_lam(prev, next);
            continue;
          } 
          else if (ptag == APP && tag == SUP) {
            next = d_ic_app_sup(prev, next); 
            continue;
          }
          else if ((ptag == CO0 || ptag == CO1) && tag == LAM) {
            next = d_ic_col_lam(prev, next);
            continue;
          }
          else if ((ptag == CO0 || ptag == CO1) && tag == SUP) {
            next = d_ic_col_sup(prev, next);
            continue;
          }
          
          // No interaction found, proceed to stack traversal
          d_stack[stack_pos++] = prev;
          break;
        }
      }
    }

    // After processing, check stack and update heap if needed
    if (stack_pos == stop) {
      d_stack_pos = stack_pos;
      return next; // Stack empty, return WHNF
    } else {
      while (stack_pos > stop) {
        Term host = d_stack[--stack_pos];
        TermTag htag = TERM_TAG(host);
        uint32_t hloc = TERM_VAL(host);
        
        // Update the heap with the reduced term
        if (htag == APP || htag == CO0 || htag == CO1) {
          d_heap[hloc] = next;
        }
        next = host;
      }
      d_stack_pos = stack_pos;
      return next; // Return updated original term
    }
  }
}

// Reduce a term to normal form
__device__ inline Term d_ic_normal(Term term) {
  // Reset stack
  d_stack_pos = 0;
  uint32_t stack_pos = 0;

  // Allocate a new node for the initial term
  uint32_t root_loc = d_ic_alloc(1);
  d_heap[root_loc] = term;

  // Push initial location to stack as a "location"
  d_stack[stack_pos++] = MAKE_TERM(false, 0, 0, root_loc);

  while (stack_pos > 0) {
    // Pop current location from stack
    uint32_t loc = TERM_VAL(d_stack[--stack_pos]);

    // Get term at this location
    Term current = d_heap[loc];

    // Reduce to WHNF
    d_stack_pos = stack_pos;
    current = d_ic_whnf(current);
    stack_pos = d_stack_pos;

    // Store the WHNF term back to the heap
    d_heap[loc] = current;

    // Get term details
    TermTag tag = TERM_TAG(current);
    uint32_t val = TERM_VAL(current);

    // Push subterm locations based on term type
    if (tag == LAM) {
      d_stack[stack_pos++] = MAKE_TERM(false, 0, 0, val);
    }
    else if (tag == APP || tag == SUP) {
      // Both APP and SUP need to push two locations
      d_stack[stack_pos++] = MAKE_TERM(false, 0, 0, val);
      d_stack[stack_pos++] = MAKE_TERM(false, 0, 0, val + 1);
    }
    // Other tags have no subterms to process
  }

  // Update stack position and return the fully normalized term
  d_stack_pos = stack_pos;
  return d_heap[root_loc];
}

// CUDA kernel to normalize a term
__global__ void normalizeKernel() {
  // Single-threaded implementation (block 0, thread 0)
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    // Get the term from the heap's entry point
    Term term = d_heap[0];
    
    // Perform normalization
    term = d_ic_normal(term);
    
    // Store the result back to the heap's entry point
    d_heap[0] = term;
  }
}

// Host function to normalize a term on the GPU
extern "C" Term ic_normal_cuda(IC* ic, Term term) {
  // Allocate GPU memory for heap
  Term* d_heap_ptr;
  Term* d_stack_ptr;
  uint32_t heap_size = ic->heap_size;
  uint32_t stack_size = ic->stack_size;
  
  // Allocate device memory for heap and stack
  hipError_t err;
  err = hipMalloc((void**)&d_heap_ptr, heap_size * sizeof(Term));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap allocation): %s\n", hipGetErrorString(err));
    return term; // Return original term on error
  }
  
  err = hipMalloc((void**)&d_stack_ptr, stack_size * sizeof(Term));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (stack allocation): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    return term; // Return original term on error
  }
  
  // Copy heap from host to device
  err = hipMemcpy(d_heap_ptr, ic->heap, ic->heap_pos * sizeof(Term), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap copy to device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    return term; // Return original term on error
  }
  
  // Set up constants on device
  uint32_t heap_pos = ic->heap_pos;
  uint64_t interactions = 0;
  uint32_t stack_pos = 0;
  
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap), &d_heap_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack), &d_stack_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_size), &heap_size, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack_size), &stack_size, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_pos), &heap_pos, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack_pos), &stack_pos, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_interactions), &interactions, sizeof(uint64_t));
  
  // Launch kernel with a single thread
  normalizeKernel<<<1, 1>>>();
  
  // Wait for kernel to complete
  hipDeviceSynchronize();
  
  // Check for kernel errors
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Kernel Error: %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    return term; // Return original term on error
  }
  
  // Get updated values back from device
  hipMemcpyFromSymbol(&heap_pos, HIP_SYMBOL(d_heap_pos), sizeof(uint32_t));
  hipMemcpyFromSymbol(&interactions, HIP_SYMBOL(d_interactions), sizeof(uint64_t));
  
  // Copy updated heap back to host
  err = hipMemcpy(ic->heap, d_heap_ptr, heap_pos * sizeof(Term), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap copy from device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    return term; // Return original term on error
  }
  
  // Update the host context
  ic->heap_pos = heap_pos;
  ic->interactions = interactions;
  
  // Free device memory
  hipFree(d_heap_ptr);
  hipFree(d_stack_ptr);
  
  // Return the normalized term
  return ic->heap[0];
}

#include "hip/hip_runtime.h"
/*

(old implementation omitted)

PROBLEM: the file above is working perfectly, but, it is only running the
runtime in a single-threaded fashion. Our goal is to update this implementation
in order to run it with N threads instead. To do so, we must hipMalloc a buffer
of 16 GB exactly (matching the target GPU memory), and, then, split it into N
buffers of 16/N GB, one for each thread. We must also copy the initial term
(i.e., the slice of the original IC object, from 0 to size), to each thread IC,
making sure each receives the same identical term. This can be done with a
kernel, where each thread copies the initial term into its own IC object. Note
that each thread will also keep its own interaction counter, which will be added
at the end to compute the total interaction count. Also, make sure to split the
stack among threads too. Ideally, the stack and the heap size should be the same,
i.e., for N=16 threads, we'd have 1 GB per thread, with a 512 MB heap and a 512
MB stack.

Refactor this file whole to make sure it uses N threads now.
Keep everything else the same.
Pay extra attention to avoid errors.
Do it now.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "ic.h"

// Check if CUDA is available
extern "C" int ic_cuda_available() {
  int deviceCount = 0;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  
  if (error != hipSuccess) {
    return 0; // CUDA error occurred
  }
  
  return deviceCount > 0;
}

// Device memory for heap, stack, and counters
__device__ Term* d_heap;
__device__ Term* d_stack;
__device__ uint32_t d_heap_size_per_thread;
__device__ uint32_t d_stack_size_per_thread;
__device__ uint32_t* d_heap_pos_array;  // Per-thread heap positions
__device__ unsigned long long* d_interactions_array;  // Per-thread interaction counters

// Device implementations of IC functions

// Create a term with substitution bit
__device__ inline Term d_ic_make_sub(Term term) {
  return term | TERM_SUB_MASK;
}

// Remove substitution bit from a term
__device__ inline Term d_ic_clear_sub(Term term) {
  return term & ~TERM_SUB_MASK;
}

// Create a term with specified tag, label, and value
__device__ inline Term d_ic_make_term(TermTag tag, uint8_t lab, uint32_t val) {
  return MAKE_TERM(false, tag, lab, val);
}

// Allocate n consecutive terms in memory (thread-safe per thread)
__device__ inline uint32_t d_ic_alloc(uint32_t n, int tid) {
  uint32_t ptr = d_heap_pos_array[tid];
  d_heap_pos_array[tid] += n;
  
  // Check if we've run out of memory for this thread
  if (d_heap_pos_array[tid] >= (tid + 1) * d_heap_size_per_thread) {
    // Wrap around within thread's heap segment (simplified error handling)
    d_heap_pos_array[tid] = (tid + 1) * d_heap_size_per_thread - 1;
  }
  
  return ptr;
}

// Apply a lambda to an argument
__device__ inline Term d_ic_app_lam(Term app, Term lam, int tid) {
  atomicAdd(&d_interactions_array[tid], 1ULL);
  
  uint32_t app_loc = TERM_VAL(app);
  uint32_t lam_loc = TERM_VAL(lam);
  
  Term arg = d_heap[app_loc + 1];
  Term bod = d_heap[lam_loc + 0];

  // Create substitution for the lambda variable
  d_heap[lam_loc] = d_ic_make_sub(arg);

  return bod;
}

// Apply a superposition
__device__ inline Term d_ic_app_sup(Term app, Term sup, int tid) {
  atomicAdd(&d_interactions_array[tid], 1ULL);
  
  uint32_t app_loc = TERM_VAL(app);
  uint32_t sup_loc = TERM_VAL(sup);
  uint8_t sup_lab = TERM_LAB(sup);

  Term arg = d_heap[app_loc + 1];
  Term lft = d_heap[sup_loc + 0];
  Term rgt = d_heap[sup_loc + 1];

  // Allocate within thread's heap segment
  uint32_t col_loc = d_ic_alloc(1, tid);
  uint32_t app1_loc = d_ic_alloc(2, tid);
  
  // Adjust locations to thread's heap segment
  col_loc += tid * d_heap_size_per_thread;
  app1_loc += tid * d_heap_size_per_thread;
  
  // Store the arg in the collapser location
  d_heap[col_loc] = arg;

  // Create CO0 and CO1 terms
  Term x0 = d_ic_make_term(CO0, sup_lab, col_loc);
  Term x1 = d_ic_make_term(CO1, sup_lab, col_loc);

  // Reuse sup_loc for app0 (adjusted for thread)
  uint32_t app0_loc = sup_loc;
  d_heap[app0_loc + 1] = x0; // lft is already in heap[app0_loc + 0]

  // Set up app1
  d_heap[app1_loc + 0] = rgt;
  d_heap[app1_loc + 1] = x1;

  // Reuse app_loc for the result superposition
  d_heap[app_loc + 0] = d_ic_make_term(APP, 0, app0_loc);
  d_heap[app_loc + 1] = d_ic_make_term(APP, 0, app1_loc);

  return d_ic_make_term(SUP, sup_lab, app_loc);
}

// Collapse a lambda
__device__ inline Term d_ic_col_lam(Term col, Term lam, int tid) {
  atomicAdd(&d_interactions_array[tid], 1ULL);
  
  uint32_t col_loc = TERM_VAL(col);
  uint32_t lam_loc = TERM_VAL(lam);
  uint8_t col_lab = TERM_LAB(col);
  uint8_t is_co0 = (TERM_TAG(col) == CO0);

  Term bod = d_heap[lam_loc + 0];

  // Batch allocate memory for efficiency within thread's segment
  uint32_t alloc_start = d_ic_alloc(5, tid);
  alloc_start += tid * d_heap_size_per_thread;
  uint32_t lam0_loc = alloc_start;
  uint32_t lam1_loc = alloc_start + 1;
  uint32_t sup_loc = alloc_start + 2; // 2 locations
  uint32_t col_new_loc = alloc_start + 4;

  // Set up the superposition
  d_heap[sup_loc + 0] = d_ic_make_term(VAR, 0, lam0_loc);
  d_heap[sup_loc + 1] = d_ic_make_term(VAR, 0, lam1_loc);

  // Replace lambda's variable with the superposition
  d_heap[lam_loc] = d_ic_make_sub(d_ic_make_term(SUP, col_lab, sup_loc));

  // Set up the new collapser
  d_heap[col_new_loc] = bod;

  // Set up new lambda bodies
  d_heap[lam0_loc] = d_ic_make_term(CO0, col_lab, col_new_loc);
  d_heap[lam1_loc] = d_ic_make_term(CO1, col_lab, col_new_loc);

  // Create and return the appropriate lambda
  if (is_co0) {
    d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(LAM, 0, lam1_loc));
    return d_ic_make_term(LAM, 0, lam0_loc);
  } else {
    d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(LAM, 0, lam0_loc));
    return d_ic_make_term(LAM, 0, lam1_loc);
  }
}

// Collapse a superposition
__device__ inline Term d_ic_col_sup(Term col, Term sup, int tid) {
  atomicAdd(&d_interactions_array[tid], 1ULL);
  
  uint32_t col_loc = TERM_VAL(col);
  uint32_t sup_loc = TERM_VAL(sup);
  uint8_t col_lab = TERM_LAB(col);
  uint8_t sup_lab = TERM_LAB(sup);
  uint8_t is_co0 = (TERM_TAG(col) == CO0);

  Term lft = d_heap[sup_loc + 0];
  Term rgt = d_heap[sup_loc + 1];

  if (col_lab == sup_lab) {
    if (is_co0) {
      d_heap[col_loc] = d_ic_make_sub(rgt);
      return lft;
    } else {
      d_heap[col_loc] = d_ic_make_sub(lft);
      return rgt;
    }
  } else {
    // Allocate within thread's segment
    uint32_t sup_start = d_ic_alloc(4, tid);
    sup_start += tid * d_heap_size_per_thread;
    uint32_t sup0_loc = sup_start;
    uint32_t sup1_loc = sup_start + 2;

    // Use existing locations as collapser locations
    uint32_t col_lft_loc = sup_loc + 0;
    uint32_t col_rgt_loc = sup_loc + 1;
    
    // Set up the first superposition (for CO0)
    d_heap[sup0_loc + 0] = d_ic_make_term(CO0, col_lab, col_lft_loc);
    d_heap[sup0_loc + 1] = d_ic_make_term(CO0, col_lab, col_rgt_loc);
    
    // Set up the second superposition (for CO1)
    d_heap[sup1_loc + 0] = d_ic_make_term(CO1, col_lab, col_lft_loc);
    d_heap[sup1_loc + 1] = d_ic_make_term(CO1, col_lab, col_rgt_loc);
    
    // Set up original collapsers to point to lft and rgt
    d_heap[col_lft_loc] = lft;
    d_heap[col_rgt_loc] = rgt;

    if (is_co0) {
      d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(SUP, sup_lab, sup1_loc));
      return d_ic_make_term(SUP, sup_lab, sup0_loc);
    } else {
      d_heap[col_loc] = d_ic_make_sub(d_ic_make_term(SUP, sup_lab, sup0_loc));
      return d_ic_make_term(SUP, sup_lab, sup1_loc);
    }
  }
}

// Reduce a term to WHNF (Weak Head Normal Form) with per-thread stack
__device__ inline Term d_ic_whnf(Term term, Term* stack, uint32_t* stack_pos, int tid) {
  uint32_t stop = *stack_pos;
  Term next = term;
  uint32_t local_stack_pos = stop;

  while (1) {
    TermTag tag = TERM_TAG(next);

    switch (tag) {
      case VAR: {
        uint32_t var_loc = TERM_VAL(next);
        Term subst = d_heap[var_loc];
        if (TERM_SUB(subst)) {
          next = d_ic_clear_sub(subst);
          continue;
        }
        break;
      }

      case CO0:
      case CO1: {
        uint32_t col_loc = TERM_VAL(next);
        Term val = d_heap[col_loc];
        if (TERM_SUB(val)) {
          next = d_ic_clear_sub(val);
          continue;
        } else {
          stack[local_stack_pos++] = next;
          next = val;
          continue;
        }
      }

      case APP: {
        uint32_t app_loc = TERM_VAL(next);
        stack[local_stack_pos++] = next;
        next = d_heap[app_loc];
        continue;
      }

      default: {
        if (local_stack_pos == stop) {
          *stack_pos = local_stack_pos;
          return next;
        } else {
          Term prev = stack[--local_stack_pos];
          TermTag ptag = TERM_TAG(prev);
          
          if (ptag == APP && tag == LAM) {
            next = d_ic_app_lam(prev, next, tid);
            continue;
          } 
          else if (ptag == APP && tag == SUP) {
            next = d_ic_app_sup(prev, next, tid);
            continue;
          }
          else if ((ptag == CO0 || ptag == CO1) && tag == LAM) {
            next = d_ic_col_lam(prev, next, tid);
            continue;
          }
          else if ((ptag == CO0 || ptag == CO1) && tag == SUP) {
            next = d_ic_col_sup(prev, next, tid);
            continue;
          }
          
          stack[local_stack_pos++] = prev;
          break;
        }
      }
    }

    if (local_stack_pos == stop) {
      *stack_pos = local_stack_pos;
      return next;
    } else {
      while (local_stack_pos > stop) {
        Term host = stack[--local_stack_pos];
        TermTag htag = TERM_TAG(host);
        uint32_t hloc = TERM_VAL(host);
        
        if (htag == APP || htag == CO0 || htag == CO1) {
          d_heap[hloc] = next;
        }
        next = host;
      }
      *stack_pos = local_stack_pos;
      return next;
    }
  }
}

// Reduce a term to normal form with per-thread stack
__device__ inline Term d_ic_normal(Term term, Term* stack, uint32_t* stack_pos, int tid) {
  // Reset stack
  *stack_pos = 0;
  uint32_t local_stack_pos = 0;

  // Allocate a new node for the initial term within thread's heap
  uint32_t root_loc = d_ic_alloc(1, tid) + tid * d_heap_size_per_thread;
  d_heap[root_loc] = term;

  // Push initial location to stack as a "location"
  stack[local_stack_pos++] = MAKE_TERM(false, 0, 0, root_loc);

  while (local_stack_pos > 0) {
    // Pop current location from stack
    uint32_t loc = TERM_VAL(stack[--local_stack_pos]);

    // Get term at this location
    Term current = d_heap[loc];

    // Reduce to WHNF
    current = d_ic_whnf(current, stack, &local_stack_pos, tid);

    // Store the WHNF term back to the heap
    d_heap[loc] = current;

    // Get term details
    TermTag tag = TERM_TAG(current);
    uint32_t val = TERM_VAL(current);

    // Push subterm locations based on term type
    if (tag == LAM) {
      stack[local_stack_pos++] = MAKE_TERM(false, 0, 0, val);
    }
    else if (tag == APP || tag == SUP) {
      stack[local_stack_pos++] = MAKE_TERM(false, 0, 0, val);
      stack[local_stack_pos++] = MAKE_TERM(false, 0, 0, val + 1);
    }
  }

  *stack_pos = local_stack_pos;
  return d_heap[root_loc];
}

// CUDA kernel to copy initial term and normalize
__global__ void normalizeKernel(int N, uint32_t initial_size) {
  int tid = threadIdx.x;
  if (tid >= N) return;

  // Define thread-specific heap and stack offsets
  uint32_t heap_offset = tid * d_heap_size_per_thread;
  uint32_t stack_offset = tid * d_stack_size_per_thread;
  Term* thread_stack = d_stack + stack_offset;
  uint32_t thread_stack_pos = 0;

  // Copy initial term to thread's heap segment
  for (uint32_t i = 0; i < initial_size; i++) {
    d_heap[heap_offset + i] = d_heap[i];
  }
  d_heap_pos_array[tid] = initial_size;

  // Normalize the term in thread's heap segment
  Term term = d_heap[heap_offset];
  term = d_ic_normal(term, thread_stack, &thread_stack_pos, tid);
  d_heap[heap_offset] = term;
}

// Host function to normalize a term on the GPU with N threads
extern "C" Term ic_normal_cuda(IC* ic, Term term) {
  // Total buffer size: 16 GB
  const size_t TOTAL_BUFFER_SIZE = 16ULL * 1024ULL * 1024ULL * 1024ULL; // 16 GB in bytes
  const int N = 16; // Number of threads (e.g., 16 as suggested)
  const size_t BUFFER_SIZE_PER_THREAD = TOTAL_BUFFER_SIZE / N; // 1 GB per thread
  const size_t HEAP_SIZE_PER_THREAD_BYTES = BUFFER_SIZE_PER_THREAD / 2; // 512 MB heap
  const size_t STACK_SIZE_PER_THREAD_BYTES = BUFFER_SIZE_PER_THREAD / 2; // 512 MB stack
  const uint32_t heap_size_per_thread = HEAP_SIZE_PER_THREAD_BYTES / sizeof(Term);
  const uint32_t stack_size_per_thread = STACK_SIZE_PER_THREAD_BYTES / sizeof(Term);

  // Allocate device memory
  Term* d_heap_ptr;
  Term* d_stack_ptr;
  uint32_t* d_heap_pos_array_ptr;
  unsigned long long* d_interactions_array_ptr;
  
  hipError_t err;
  err = hipMalloc((void**)&d_heap_ptr, N * heap_size_per_thread * sizeof(Term));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap allocation): %s\n", hipGetErrorString(err));
    return term;
  }
  
  err = hipMalloc((void**)&d_stack_ptr, N * stack_size_per_thread * sizeof(Term));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (stack allocation): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    return term;
  }
  
  err = hipMalloc((void**)&d_heap_pos_array_ptr, N * sizeof(uint32_t));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap pos array allocation): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    return term;
  }
  
  err = hipMalloc((void**)&d_interactions_array_ptr, N * sizeof(unsigned long long));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (interactions array allocation): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    return term;
  }

  // Copy initial heap to device (to first thread's segment initially)
  err = hipMemcpy(d_heap_ptr, ic->heap, ic->heap_pos * sizeof(Term), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap copy to device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    return term;
  }

  // Initialize heap positions and interaction counters
  uint32_t* h_heap_pos_array = (uint32_t*)malloc(N * sizeof(uint32_t));
  unsigned long long* h_interactions_array = (unsigned long long*)malloc(N * sizeof(unsigned long long));
  for (int i = 0; i < N; i++) {
    h_heap_pos_array[i] = ic->heap_pos;
    h_interactions_array[i] = 0;
  }
  hipMemcpy(d_heap_pos_array_ptr, h_heap_pos_array, N * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_interactions_array_ptr, h_interactions_array, N * sizeof(unsigned long long), hipMemcpyHostToDevice);

  // Set up device constants
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap), &d_heap_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack), &d_stack_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_size_per_thread), &heap_size_per_thread, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack_size_per_thread), &stack_size_per_thread, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_pos_array), &d_heap_pos_array_ptr, sizeof(uint32_t*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_interactions_array), &d_interactions_array_ptr, sizeof(unsigned long long*));

  // Launch kernel with N threads
  normalizeKernel<<<1, N>>>(N, ic->heap_pos);

  // Wait for kernel to complete
  hipDeviceSynchronize();

  // Check for kernel errors
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Kernel Error: %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    free(h_heap_pos_array);
    free(h_interactions_array);
    return term;
  }

  // Copy back heap positions and interaction counters
  hipMemcpy(h_heap_pos_array, d_heap_pos_array_ptr, N * sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(h_interactions_array, d_interactions_array_ptr, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);

  // Sum interactions and find max heap position
  uint64_t total_interactions = 0;
  uint32_t max_heap_pos = 0;
  for (int i = 0; i < N; i++) {
    total_interactions += h_interactions_array[i];
    if (h_heap_pos_array[i] > max_heap_pos) {
      max_heap_pos = h_heap_pos_array[i];
    }
  }

  // Copy normalized heap back from thread 0's segment
  err = hipMemcpy(ic->heap, d_heap_ptr, max_heap_pos * sizeof(Term), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap copy from device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    free(h_heap_pos_array);
    free(h_interactions_array);
    return term;
  }

  // Update host context
  ic->heap_pos = max_heap_pos;
  ic->interactions = total_interactions;

  // Free device memory
  hipFree(d_heap_ptr);
  hipFree(d_stack_ptr);
  hipFree(d_heap_pos_array_ptr);
  hipFree(d_interactions_array_ptr);
  free(h_heap_pos_array);
  free(h_interactions_array);

  // Return the normalized term from thread 0
  return ic->heap[0];
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "ic.h"

// Maximum threads per block (hardware constraint)
#define IC_MAX_THREADS_PER_BLOCK 512

// Check if CUDA is available
extern "C" int ic_cuda_available() {
  int deviceCount = 0;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) {
    return 0; // CUDA error occurred
  }
  return deviceCount > 0;
}

// Device memory pointers and sizes
__device__ Term* d_heap;
__device__ Term* d_stack;
__device__ uint8_t* d_subst;                   // Substitution bitmap
__device__ uint32_t d_heap_size_per_thread;
__device__ uint32_t d_stack_size_per_thread;
__device__ uint32_t d_subst_size_per_thread;   // Size of bitmap in bytes
__device__ uint32_t* d_heap_pos_array;          // Per-thread heap positions
__device__ unsigned long long* d_interactions_array; // Per-thread interaction counters

// Device utility functions

// Set the substitution bit for a location
__device__ inline void d_ic_set_subst(uint32_t loc, uint8_t* subst) {
  uint32_t byte_idx = loc / 8;
  uint8_t bit_idx = loc % 8;
  // Use atomicOr for thread safety
  atomicOr(&subst[byte_idx], 1 << bit_idx);
}

// Check if a location is marked as a substitution
__device__ inline bool d_ic_is_subst(uint32_t loc, uint8_t* subst) {
  uint32_t byte_idx = loc / 8;
  uint8_t bit_idx = loc % 8;
  return (subst[byte_idx] & (1 << bit_idx)) != 0;
}

// Clear the substitution bit for a location
__device__ inline void d_ic_clear_subst(uint32_t loc, uint8_t* subst) {
  uint32_t byte_idx = loc / 8;
  uint8_t bit_idx = loc % 8;
  // Use atomicAnd for thread safety
  atomicAnd(&subst[byte_idx], ~(1 << bit_idx));
}

// Create a term with specified tag and value
__device__ inline Term d_ic_make_term(TermTag tag, uint32_t val) {
  return MAKE_TERM(tag, val);
}

// Create a superposition term with the given label
__device__ inline Term d_ic_make_sup(uint8_t label, uint32_t val) {
  return d_ic_make_term((TermTag)(SP0 + (label & 0x3)), val);
}

// Create a collapser X term with the given label
__device__ inline Term d_ic_make_col_x(uint8_t label, uint32_t val) {
  return d_ic_make_term((TermTag)(CX0 + (label & 0x3)), val);
}

// Create a collapser Y term with the given label
__device__ inline Term d_ic_make_col_y(uint8_t label, uint32_t val) {
  return d_ic_make_term((TermTag)(CY0 + (label & 0x3)), val);
}

// Get the label from a term tag
__device__ inline uint8_t d_ic_get_label(TermTag tag) {
  if (tag >= SP0 && tag <= SP3) {
    return tag - SP0;
  } else if (tag >= CX0 && tag <= CY3) {
    return (tag - CX0) & 0x3;
  }
  return 0;
}

// Check if a term tag is a collapser X
__device__ inline bool d_ic_is_col_x(TermTag tag) {
  return (tag >= CX0 && tag <= CX3);
}

// Check if a term tag is a collapser Y
__device__ inline bool d_ic_is_col_y(TermTag tag) {
  return (tag >= CY0 && tag <= CY3);
}

// Check if a term tag is a superposition
__device__ inline bool d_ic_is_sup(TermTag tag) {
  return (tag >= SP0 && tag <= SP3);
}

// Allocate n consecutive terms in the thread's local heap
__device__ inline uint32_t d_ic_alloc(uint32_t n, uint32_t* heap_pos) {
  if (*heap_pos + n > d_heap_size_per_thread) {
    // Handle overflow - return current position without updating
    // This prevents illegal memory access but may cause computation errors
    return *heap_pos;
  }
  
  uint32_t ptr = *heap_pos;
  *heap_pos += n;
  return ptr;
}

// Apply a lambda to an argument
__device__ inline Term d_ic_app_lam(Term app, Term lam, Term* heap, uint8_t* subst, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t app_loc = TERM_VAL(app);
  uint32_t lam_loc = TERM_VAL(lam);
  Term arg = heap[app_loc + 1];
  Term bod = heap[lam_loc + 0];
  heap[lam_loc] = arg;
  d_ic_set_subst(lam_loc, subst);
  return bod;
}

// Apply a superposition
__device__ inline Term d_ic_app_sup(Term app, Term sup, Term* heap, uint8_t* subst, uint32_t* heap_pos, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t app_loc = TERM_VAL(app);
  uint32_t sup_loc = TERM_VAL(sup);
  TermTag sup_tag = TERM_TAG(sup);
  uint8_t sup_lab = d_ic_get_label(sup_tag);
  Term arg = heap[app_loc + 1];
  Term lft = heap[sup_loc + 0];
  Term rgt = heap[sup_loc + 1];

  uint32_t col_loc = d_ic_alloc(1, heap_pos);
  uint32_t app1_loc = d_ic_alloc(2, heap_pos);
  heap[col_loc] = arg;

  Term x0 = d_ic_make_col_x(sup_lab, col_loc);
  Term x1 = d_ic_make_col_y(sup_lab, col_loc);

  heap[sup_loc + 1] = x0; // Reuse sup_loc as app0_loc
  heap[app1_loc + 0] = rgt;
  heap[app1_loc + 1] = x1;

  heap[app_loc + 0] = d_ic_make_term(APP, sup_loc);
  heap[app_loc + 1] = d_ic_make_term(APP, app1_loc);
  return d_ic_make_sup(sup_lab, app_loc);
}

// Collapse a lambda
__device__ inline Term d_ic_col_lam(Term col, Term lam, Term* heap, uint8_t* subst, uint32_t* heap_pos, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t col_loc = TERM_VAL(col);
  uint32_t lam_loc = TERM_VAL(lam);
  TermTag col_tag = TERM_TAG(col);
  uint8_t col_lab = d_ic_get_label(col_tag);
  bool is_col_x = d_ic_is_col_x(col_tag);
  Term bod = heap[lam_loc + 0];

  uint32_t alloc_start = d_ic_alloc(5, heap_pos);
  uint32_t lam0_loc = alloc_start;
  uint32_t lam1_loc = alloc_start + 1;
  uint32_t sup_loc = alloc_start + 2;
  uint32_t col_new_loc = alloc_start + 4;

  heap[sup_loc + 0] = d_ic_make_term(VAR, lam0_loc);
  heap[sup_loc + 1] = d_ic_make_term(VAR, lam1_loc);
  heap[lam_loc] = d_ic_make_sup(col_lab, sup_loc);
  d_ic_set_subst(lam_loc, subst);
  heap[col_new_loc] = bod;
  heap[lam0_loc] = d_ic_make_col_x(col_lab, col_new_loc);
  heap[lam1_loc] = d_ic_make_col_y(col_lab, col_new_loc);

  if (is_col_x) {
    heap[col_loc] = d_ic_make_term(LAM, lam1_loc);
    d_ic_set_subst(col_loc, subst);
    return d_ic_make_term(LAM, lam0_loc);
  } else {
    heap[col_loc] = d_ic_make_term(LAM, lam0_loc);
    d_ic_set_subst(col_loc, subst);
    return d_ic_make_term(LAM, lam1_loc);
  }
}

// Collapse a superposition
__device__ inline Term d_ic_col_sup(Term col, Term sup, Term* heap, uint8_t* subst, uint32_t* heap_pos, unsigned long long* interactions) {
  atomicAdd(interactions, 1ULL);
  uint32_t col_loc = TERM_VAL(col);
  uint32_t sup_loc = TERM_VAL(sup);
  TermTag col_tag = TERM_TAG(col);
  TermTag sup_tag = TERM_TAG(sup);
  uint8_t col_lab = d_ic_get_label(col_tag);
  uint8_t sup_lab = d_ic_get_label(sup_tag);
  bool is_col_x = d_ic_is_col_x(col_tag);
  Term lft = heap[sup_loc + 0];
  Term rgt = heap[sup_loc + 1];

  if (col_lab == sup_lab) {
    if (is_col_x) {
      heap[col_loc] = rgt;
      d_ic_set_subst(col_loc, subst);
      return lft;
    } else {
      heap[col_loc] = lft;
      d_ic_set_subst(col_loc, subst);
      return rgt;
    }
  } else {
    uint32_t sup_start = d_ic_alloc(4, heap_pos);
    uint32_t sup0_loc = sup_start;
    uint32_t sup1_loc = sup_start + 2;

    heap[sup0_loc + 0] = d_ic_make_col_x(col_lab, sup_loc + 0);
    heap[sup0_loc + 1] = d_ic_make_col_x(col_lab, sup_loc + 1);
    heap[sup1_loc + 0] = d_ic_make_col_y(col_lab, sup_loc + 0);
    heap[sup1_loc + 1] = d_ic_make_col_y(col_lab, sup_loc + 1);
    heap[sup_loc + 0] = lft;
    heap[sup_loc + 1] = rgt;

    if (is_col_x) {
      heap[col_loc] = d_ic_make_sup(sup_lab, sup1_loc);
      d_ic_set_subst(col_loc, subst);
      return d_ic_make_sup(sup_lab, sup0_loc);
    } else {
      heap[col_loc] = d_ic_make_sup(sup_lab, sup0_loc);
      d_ic_set_subst(col_loc, subst);
      return d_ic_make_sup(sup_lab, sup1_loc);
    }
  }
}

// Reduce a term to WHNF (Weak Head Normal Form)
__device__ inline Term d_ic_whnf(Term term, Term* heap, uint8_t* subst, Term* stack, uint32_t* heap_pos, uint32_t* stack_pos, unsigned long long* interactions) {
  uint32_t stop = *stack_pos;
  Term next = term;
  uint32_t local_stack_pos = stop;

  while (true) {
    TermTag tag = TERM_TAG(next);
    switch (tag) {
      case VAR: {
        uint32_t var_loc = TERM_VAL(next);
        if (d_ic_is_subst(var_loc, subst)) {
          next = heap[var_loc];
          continue;
        }
        break;
      }
      case CX0:
      case CY0:
      case CX1:
      case CY1:
      case CX2:
      case CY2:
      case CX3:
      case CY3: {
        uint32_t col_loc = TERM_VAL(next);
        if (d_ic_is_subst(col_loc, subst)) {
          next = heap[col_loc];
          continue;
        } else {
          stack[local_stack_pos++] = next;
          next = heap[col_loc];
          continue;
        }
      }
      case APP: {
        uint32_t app_loc = TERM_VAL(next);
        stack[local_stack_pos++] = next;
        next = heap[app_loc];
        continue;
      }
      default: {
        if (local_stack_pos == stop) {
          *stack_pos = local_stack_pos;
          return next;
        }
        Term prev = stack[--local_stack_pos];
        TermTag ptag = TERM_TAG(prev);
        if (ptag == APP && tag == LAM) {
          next = d_ic_app_lam(prev, next, heap, subst, interactions);
          continue;
        } else if (ptag == APP && (tag >= SP0 && tag <= SP3)) {
          next = d_ic_app_sup(prev, next, heap, subst, heap_pos, interactions);
          continue;
        } else if ((ptag >= CX0 && ptag <= CY3) && tag == LAM) {
          next = d_ic_col_lam(prev, next, heap, subst, heap_pos, interactions);
          continue;
        } else if ((ptag >= CX0 && ptag <= CY3) && (tag >= SP0 && tag <= SP3)) {
          next = d_ic_col_sup(prev, next, heap, subst, heap_pos, interactions);
          continue;
        }
        stack[local_stack_pos++] = prev;
        break;
      }
    }
    if (local_stack_pos == stop) {
      *stack_pos = local_stack_pos;
      return next;
    }
    while (local_stack_pos > stop) {
      Term host = stack[--local_stack_pos];
      TermTag htag = TERM_TAG(host);
      uint32_t hloc = TERM_VAL(host);
      if (htag == APP || (htag >= CX0 && htag <= CY3)) {
        heap[hloc] = next;
      }
      next = host;
    }
    *stack_pos = local_stack_pos;
    return next;
  }
}

// Reduce a term to normal form
__device__ inline Term d_ic_normal(Term term, Term* heap, uint8_t* subst, Term* stack, uint32_t* heap_pos, uint32_t* stack_pos, unsigned long long* interactions) {
  *stack_pos = 0;
  uint32_t local_stack_pos = 0;
  uint32_t root_loc = d_ic_alloc(1, heap_pos);
  heap[root_loc] = term;
  stack[local_stack_pos++] = d_ic_make_term(VAR, root_loc);

  while (local_stack_pos > 0) {
    uint32_t loc = TERM_VAL(stack[--local_stack_pos]);
    Term current = heap[loc];
    current = d_ic_whnf(current, heap, subst, stack, heap_pos, &local_stack_pos, interactions);
    heap[loc] = current;
    TermTag tag = TERM_TAG(current);
    uint32_t val = TERM_VAL(current);
    if (tag == LAM) {
      stack[local_stack_pos++] = d_ic_make_term(VAR, val);
    } else if (tag == APP || (tag >= SP0 && tag <= SP3)) {
      stack[local_stack_pos++] = d_ic_make_term(VAR, val);
      stack[local_stack_pos++] = d_ic_make_term(VAR, val + 1);
    }
  }
  *stack_pos = local_stack_pos;
  return heap[root_loc];
}

// CUDA kernel for normalization
__global__ void normalizeKernel(int N, uint32_t initial_size) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= N) return;

  // Local heap, stack, and substitution bitmap for this thread
  Term* local_heap = d_heap + (tid * d_heap_size_per_thread);
  Term* local_stack = d_stack + (tid * d_stack_size_per_thread);
  uint8_t* local_subst = d_subst + (tid * d_subst_size_per_thread);
  uint32_t* local_heap_pos = &d_heap_pos_array[tid];
  uint32_t local_stack_pos = 0;
  unsigned long long* local_interactions = &d_interactions_array[tid];

  // Initialize substitution bitmap to zeros
  for (uint32_t i = 0; i < d_subst_size_per_thread; i++) {
    local_subst[i] = 0;
  }

  // Copy initial term to local heap
  for (uint32_t i = 0; i < initial_size; i++) {
    Term original = d_heap[i];
    TermTag tag = TERM_TAG(original);
    uint32_t val = TERM_VAL(original);
    local_heap[i] = d_ic_make_term(tag, val);
  }
  *local_heap_pos = initial_size;

  // Normalize the term
  Term term = local_heap[0];
  term = d_ic_normal(term, local_heap, local_subst, local_stack, local_heap_pos, &local_stack_pos, local_interactions);
  local_heap[0] = term;
}

// Host function to normalize a term on the GPU
extern "C" Term ic_normal_cuda(IC* ic, Term term, int thread_count) {
  const size_t TOTAL_BUFFER_SIZE = 22ULL * 1024ULL * 1024ULL * 1024ULL; // 22 GB
  const size_t HEAP_PORTION = 16ULL * 1024ULL * 1024ULL * 1024ULL; // 16 GB for heaps
  const size_t STACK_PORTION = 4ULL * 1024ULL * 1024ULL * 1024ULL; // 4 GB for stacks
  const size_t SUBST_PORTION = 2ULL * 1024ULL * 1024ULL * 1024ULL; // 2 GB for substitution bitmaps
  
  int N = (thread_count > 0) ? thread_count : 1;
  
  const size_t HEAP_SIZE_PER_THREAD_BYTES = HEAP_PORTION / N;
  const size_t STACK_SIZE_PER_THREAD_BYTES = STACK_PORTION / N;
  const size_t SUBST_SIZE_PER_THREAD_BYTES = SUBST_PORTION / N;
  
  const uint32_t heap_size_per_thread = HEAP_SIZE_PER_THREAD_BYTES / sizeof(Term);
  const uint32_t stack_size_per_thread = STACK_SIZE_PER_THREAD_BYTES / sizeof(Term);
  const uint32_t subst_size_per_thread = SUBST_SIZE_PER_THREAD_BYTES / sizeof(uint8_t);

  // Display memory allocation info for debug purposes
  printf("Memory Config: Total: %.2f GB (Heap: %.2f GB, Stack: %.2f GB, Subst: %.2f GB)\n", 
         TOTAL_BUFFER_SIZE / (1024.0 * 1024.0 * 1024.0),
         HEAP_PORTION / (1024.0 * 1024.0 * 1024.0),
         STACK_PORTION / (1024.0 * 1024.0 * 1024.0),
         SUBST_PORTION / (1024.0 * 1024.0 * 1024.0));
  printf("Per Thread: Heap: %.2f MB, Stack: %.2f MB, Subst: %.2f MB\n",
         HEAP_SIZE_PER_THREAD_BYTES / (1024.0 * 1024.0),
         STACK_SIZE_PER_THREAD_BYTES / (1024.0 * 1024.0),
         SUBST_SIZE_PER_THREAD_BYTES / (1024.0 * 1024.0));

  // Device memory allocation
  Term *d_heap_ptr, *d_stack_ptr;
  uint8_t *d_subst_ptr;
  uint32_t* d_heap_pos_array_ptr;
  unsigned long long* d_interactions_array_ptr;
  hipError_t err;

  err = hipMalloc(&d_heap_ptr, HEAP_PORTION);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap): %s\n", hipGetErrorString(err));
    return term;
  }

  err = hipMalloc(&d_stack_ptr, STACK_PORTION);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (stack): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    return term;
  }

  err = hipMalloc(&d_subst_ptr, SUBST_PORTION);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (subst): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    return term;
  }

  err = hipMalloc(&d_heap_pos_array_ptr, N * sizeof(uint32_t));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap_pos): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_subst_ptr);
    return term;
  }

  err = hipMalloc(&d_interactions_array_ptr, N * sizeof(unsigned long long));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (interactions): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_subst_ptr);
    hipFree(d_heap_pos_array_ptr);
    return term;
  }

  // Copy initial heap to device (temporary buffer for kernel to distribute)
  err = hipMemcpy(d_heap_ptr, ic->heap, ic->heap_pos * sizeof(Term), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (copy to device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_subst_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    return term;
  }

  // Initialize arrays
  uint32_t* h_heap_pos_array = (uint32_t*)malloc(N * sizeof(uint32_t));
  unsigned long long* h_interactions_array = (unsigned long long*)malloc(N * sizeof(unsigned long long));
  for (int i = 0; i < N; i++) {
    h_heap_pos_array[i] = 0;
    h_interactions_array[i] = 0;
  }
  hipMemcpy(d_heap_pos_array_ptr, h_heap_pos_array, N * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_interactions_array_ptr, h_interactions_array, N * sizeof(unsigned long long), hipMemcpyHostToDevice);

  // Set device symbols
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap), &d_heap_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack), &d_stack_ptr, sizeof(Term*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_subst), &d_subst_ptr, sizeof(uint8_t*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_size_per_thread), &heap_size_per_thread, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_stack_size_per_thread), &stack_size_per_thread, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_subst_size_per_thread), &subst_size_per_thread, sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_heap_pos_array), &d_heap_pos_array_ptr, sizeof(uint32_t*));
  hipMemcpyToSymbol(HIP_SYMBOL(d_interactions_array), &d_interactions_array_ptr, sizeof(unsigned long long*));

  // Launch kernel
  int blocks = (N + IC_MAX_THREADS_PER_BLOCK - 1) / IC_MAX_THREADS_PER_BLOCK;
  int threads_per_block = (N < IC_MAX_THREADS_PER_BLOCK) ? N : IC_MAX_THREADS_PER_BLOCK;
  printf("CUDA Configuration: %d thread(s) total, %d block(s), %d thread(s) per block\n", N, blocks, threads_per_block);
  normalizeKernel<<<blocks, threads_per_block>>>(N, ic->heap_pos);
  hipDeviceSynchronize();

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Kernel Error: %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_subst_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    free(h_heap_pos_array);
    free(h_interactions_array);
    return term;
  }

  // Copy results back
  hipMemcpy(h_heap_pos_array, d_heap_pos_array_ptr, N * sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(h_interactions_array, d_interactions_array_ptr, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);

  uint64_t total_interactions = 0;
  for (int i = 0; i < N; i++) {
    total_interactions += h_interactions_array[i];
  }
  uint32_t heap_pos_thread0 = h_heap_pos_array[0];

  // Only copy back the result from the first thread
  err = hipMemcpy(ic->heap, d_heap_ptr, heap_pos_thread0 * sizeof(Term), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (copy from device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipFree(d_subst_ptr);
    hipFree(d_heap_pos_array_ptr);
    hipFree(d_interactions_array_ptr);
    free(h_heap_pos_array);
    free(h_interactions_array);
    return term;
  }

  ic->heap_pos = heap_pos_thread0;
  ic->interactions = total_interactions;

  hipFree(d_heap_ptr);
  hipFree(d_stack_ptr);
  hipFree(d_subst_ptr);
  hipFree(d_heap_pos_array_ptr);
  hipFree(d_interactions_array_ptr);
  free(h_heap_pos_array);
  free(h_interactions_array);

  return ic->heap[0];
}

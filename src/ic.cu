#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "ic.h"

// Check if CUDA is available
extern "C" int ic_cuda_available() {
  int deviceCount = 0;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  
  if (error != hipSuccess) {
    return 0; // CUDA error occurred
  }
  
  return deviceCount > 0;
}

// Device constants optimized for GPU performance
// Use compiler flags to direct GPU optimizations
#pragma GCC push_options
#pragma GCC optimize ("O3", "unroll-loops", "fast-math")
#define USE_AGGRESSIVE_OPTIMIZATIONS 1

// Device memory for heap and stack
__device__ Term* d_heap;
__device__ Term* d_stack;
__device__ uint32_t d_heap_size;
__device__ uint32_t d_stack_size;
__device__ uint32_t d_heap_pos;
__device__ uint32_t d_stack_pos;
__device__ uint64_t d_interactions;

// Device implementations of IC functions

// Fast, optimized version using direct bit operations
// Use direct macro expansion for critical operations
#define D_IC_MAKE_SUB(term) ((term) | TERM_SUB_MASK)
#define D_IC_CLEAR_SUB(term) ((term) & ~TERM_SUB_MASK)
#define D_IC_GET_TAG(term) ((TermTag)(((term) & TERM_TAG_MASK) >> 28))
#define D_IC_GET_LAB(term) (((term) & TERM_LAB_MASK) >> 26)
#define D_IC_GET_VAL(term) ((term) & TERM_VAL_MASK)

// Function versions maintained for compatibility
__device__ __forceinline__ Term d_ic_make_sub(Term term) {
  return D_IC_MAKE_SUB(term);
}

// Function versions maintained for compatibility
__device__ __forceinline__ Term d_ic_clear_sub(Term term) {
  return D_IC_CLEAR_SUB(term);
}

// Create a term with specified tag, label, and value
// Define as a macro for better compiler optimization
#define D_IC_MAKE_TERM(tag, lab, val) \
  (((uint32_t)(tag) << 28) | ((uint32_t)(lab) << 26) | ((uint32_t)(val) & TERM_VAL_MASK))

// Function version maintained for compatibility
__device__ __forceinline__ Term d_ic_make_term(TermTag tag, uint8_t lab, uint32_t val) {
  // Direct bit manipulation for faster term construction
  return D_IC_MAKE_TERM(tag, lab, val);
}

// Allocate n consecutive terms in memory with prefetch hint
__device__ __forceinline__ uint32_t d_ic_alloc(uint32_t n) {
  uint32_t ptr = d_heap_pos;
  d_heap_pos += n;
  
  // Check if we've run out of memory (bounds check)
  if (d_heap_pos >= d_heap_size) {
    // Since we can't easily abort a kernel, just cap at maximum size
    // This is just a safeguard; the host should ensure enough memory
    d_heap_pos = d_heap_size - 1;
  }
  
  // Prefetch next allocation area to help cache locality
  #pragma unroll
  for (uint32_t i = 0; i < 4 && i < n; i++) {
    // Use simple memory access as prefetch hint
    volatile Term temp = d_heap[ptr + i];
  }
  
  return ptr;
}

// Apply a lambda to an argument - optimized with direct heap access and macros
__device__ __forceinline__ Term d_ic_app_lam(Term app, Term lam) {
  d_interactions++;
  
  // Extract locations with optimized macros
  const uint32_t app_loc = D_IC_GET_VAL(app);
  const uint32_t lam_loc = D_IC_GET_VAL(lam);
  
  // Get heap pointer for direct access
  Term* const heap = d_heap;
  
  // Load arguments directly
  const Term arg = heap[app_loc + 1];
  const Term bod = heap[lam_loc + 0];

  // Create substitution for the lambda variable with direct bit manipulation
  heap[lam_loc] = D_IC_MAKE_SUB(arg);

  return bod;
}

// Apply a superposition - optimized
__device__ __forceinline__ Term d_ic_app_sup(Term app, Term sup) {
  d_interactions++;
  
  // Cache frequent values in registers for faster access
  const uint32_t app_loc = TERM_VAL(app);
  const uint32_t sup_loc = TERM_VAL(sup);
  const uint8_t sup_lab = TERM_LAB(sup);
  
  // Direct heap access
  Term* const heap = d_heap;

  // Load arguments in one go to reduce memory reads
  const Term arg = heap[app_loc + 1];
  const Term lft = heap[sup_loc + 0];
  const Term rgt = heap[sup_loc + 1];

  // Batch allocation for better memory access pattern
  const uint32_t col_loc = d_ic_alloc(1);
  const uint32_t app1_loc = d_ic_alloc(2);
  
  // Store the arg in the collapser location
  heap[col_loc] = arg;

  // Create terms with direct bit manipulation for speed
  // CO0 term - optimized term creation
  const Term x0 = ((uint32_t)(CO0) << 28) | ((uint32_t)(sup_lab) << 26) | (col_loc & TERM_VAL_MASK);
  
  // CO1 term - optimized term creation
  const Term x1 = ((uint32_t)(CO1) << 28) | ((uint32_t)(sup_lab) << 26) | (col_loc & TERM_VAL_MASK);

  // Reuse sup_loc for app0 - lft is already in heap[sup_loc + 0]
  heap[sup_loc + 1] = x0;

  // Set up app1
  heap[app1_loc + 0] = rgt;
  heap[app1_loc + 1] = x1;

  // Reuse app_loc for the result superposition
  // Use direct bit manipulation for d_ic_make_term
  heap[app_loc + 0] = ((uint32_t)(APP) << 28) | (sup_loc & TERM_VAL_MASK);
  heap[app_loc + 1] = ((uint32_t)(APP) << 28) | (app1_loc & TERM_VAL_MASK);

  // Use direct bit manipulation for return value
  return ((uint32_t)(SUP) << 28) | ((uint32_t)(sup_lab) << 26) | (app_loc & TERM_VAL_MASK);
}

// Collapse a lambda - optimized for GPU
__device__ __forceinline__ Term d_ic_col_lam(Term col, Term lam) {
  d_interactions++;
  
  // Cache frequent values in registers
  const uint32_t col_loc = TERM_VAL(col);
  const uint32_t lam_loc = TERM_VAL(lam);
  const uint8_t col_lab = TERM_LAB(col);
  const uint8_t is_co0 = (TERM_TAG(col) == CO0);
  
  // Direct heap access
  Term* const heap = d_heap;

  // Load body once
  const Term bod = heap[lam_loc + 0];

  // Batch allocate memory for better memory pattern
  const uint32_t alloc_start = d_ic_alloc(5);
  const uint32_t lam0_loc = alloc_start;
  const uint32_t lam1_loc = alloc_start + 1;
  const uint32_t sup_loc = alloc_start + 2; // 2 locations
  const uint32_t col_new_loc = alloc_start + 4;

  // Set up the superposition with direct bit manipulation
  heap[sup_loc + 0] = ((uint32_t)(VAR) << 28) | (lam0_loc & TERM_VAL_MASK);
  heap[sup_loc + 1] = ((uint32_t)(VAR) << 28) | (lam1_loc & TERM_VAL_MASK);

  // Replace lambda's variable with the superposition
  // Use direct bit manipulation for nested term creation
  heap[lam_loc] = ((uint32_t)(SUP) << 28) | ((uint32_t)(col_lab) << 26) | (sup_loc & TERM_VAL_MASK) | TERM_SUB_MASK;

  // Set up the new collapser
  heap[col_new_loc] = bod;

  // Set up new lambda bodies with direct bit manipulation
  heap[lam0_loc] = ((uint32_t)(CO0) << 28) | ((uint32_t)(col_lab) << 26) | (col_new_loc & TERM_VAL_MASK);
  heap[lam1_loc] = ((uint32_t)(CO1) << 28) | ((uint32_t)(col_lab) << 26) | (col_new_loc & TERM_VAL_MASK);

  // Create and return the appropriate lambda - branch-free when possible
  // Fast path implementation using registers
  const Term lam0_term = ((uint32_t)(LAM) << 28) | (lam0_loc & TERM_VAL_MASK);
  const Term lam1_term = ((uint32_t)(LAM) << 28) | (lam1_loc & TERM_VAL_MASK);
  const Term sub_term0 = ((uint32_t)(LAM) << 28) | (lam0_loc & TERM_VAL_MASK) | TERM_SUB_MASK;
  const Term sub_term1 = ((uint32_t)(LAM) << 28) | (lam1_loc & TERM_VAL_MASK) | TERM_SUB_MASK;
  
  // Use the condition directly to avoid branching when possible
  heap[col_loc] = is_co0 ? sub_term1 : sub_term0;
  return is_co0 ? lam0_term : lam1_term;
}

// Collapse a superposition - optimized with fast paths
__device__ __forceinline__ Term d_ic_col_sup(Term col, Term sup) {
  d_interactions++;
  
  // Cache frequent values in registers
  const uint32_t col_loc = TERM_VAL(col);
  const uint32_t sup_loc = TERM_VAL(sup);
  const uint8_t col_lab = TERM_LAB(col);
  const uint8_t sup_lab = TERM_LAB(sup);
  const uint8_t is_co0 = (TERM_TAG(col) == CO0);
  
  // Direct heap access
  Term* const heap = d_heap;

  // Load values needed for both paths
  const Term lft = heap[sup_loc + 0];
  const Term rgt = heap[sup_loc + 1];

  // Fast path for matching labels (more common case) - helps branch prediction
  if (col_lab == sup_lab) {
    // Labels match: simple substitution - use direct bit manipulation
    // This is the most common case, so optimize heavily
    if (is_co0) {
      heap[col_loc] = rgt | TERM_SUB_MASK;
      return lft;
    } else {
      heap[col_loc] = lft | TERM_SUB_MASK;
      return rgt;
    }
  } else {
    // Labels don't match: create nested collapsers
    // This path is less common but still needs optimization
    const uint32_t sup_start = d_ic_alloc(4); // 2 sups with 2 terms each
    const uint32_t sup0_loc = sup_start;
    const uint32_t sup1_loc = sup_start + 2;

    // Use existing locations as collapser locations to save memory
    const uint32_t col_lft_loc = sup_loc + 0;
    const uint32_t col_rgt_loc = sup_loc + 1;
    
    // Set up the first superposition (for CO0) with direct bit manipulation
    heap[sup0_loc + 0] = ((uint32_t)(CO0) << 28) | ((uint32_t)(col_lab) << 26) | (col_lft_loc & TERM_VAL_MASK);
    heap[sup0_loc + 1] = ((uint32_t)(CO0) << 28) | ((uint32_t)(col_lab) << 26) | (col_rgt_loc & TERM_VAL_MASK);
    
    // Set up the second superposition (for CO1) with direct bit manipulation
    heap[sup1_loc + 0] = ((uint32_t)(CO1) << 28) | ((uint32_t)(col_lab) << 26) | (col_lft_loc & TERM_VAL_MASK);
    heap[sup1_loc + 1] = ((uint32_t)(CO1) << 28) | ((uint32_t)(col_lab) << 26) | (col_rgt_loc & TERM_VAL_MASK);
    
    // Set up original collapsers to point to lft and rgt
    heap[col_lft_loc] = lft;
    heap[col_rgt_loc] = rgt;

    // Prepare common terms to reduce branches
    const Term sup0_term = ((uint32_t)(SUP) << 28) | ((uint32_t)(sup_lab) << 26) | (sup0_loc & TERM_VAL_MASK);
    const Term sup1_term = ((uint32_t)(SUP) << 28) | ((uint32_t)(sup_lab) << 26) | (sup1_loc & TERM_VAL_MASK);
    const Term sub_term0 = ((uint32_t)(SUP) << 28) | ((uint32_t)(sup_lab) << 26) | (sup0_loc & TERM_VAL_MASK) | TERM_SUB_MASK;
    const Term sub_term1 = ((uint32_t)(SUP) << 28) | ((uint32_t)(sup_lab) << 26) | (sup1_loc & TERM_VAL_MASK) | TERM_SUB_MASK;

    // Use the condition directly to avoid branching when possible
    heap[col_loc] = is_co0 ? sub_term1 : sub_term0;
    return is_co0 ? sup0_term : sup1_term;
  }
}

// Shared memory cache will be dynamically allocated in the kernel
// Note that we're using a single thread, so regular global memory is fine

// Key constants for faster case switching
#define INTERACTION_APP_LAM ((APP << 3) | LAM)
#define INTERACTION_APP_SUP ((APP << 3) | SUP)

// Reduce a term to WHNF (Weak Head Normal Form) - heavily optimized with macros and prefetching
__device__ __forceinline__ Term d_ic_whnf(Term term) {
  // Cache frequently used variables in registers for faster access
  register uint32_t stop = d_stack_pos;
  register Term next = term;
  Term* const __restrict__ heap = d_heap;  // Restrict pointer for better optimization
  Term* const __restrict__ stack = d_stack;
  register uint32_t stack_pos = stop;

  // Main normalization loop
  while (1) {
    // Get tag with optimized macro
    const TermTag tag = D_IC_GET_TAG(next);

    // Use switch for better branch prediction on GPU
    switch (tag) {
      case VAR: {
        // Variable case - optimize for common path
        const uint32_t var_loc = D_IC_GET_VAL(next);
        const Term subst = heap[var_loc];
        if (subst & TERM_SUB_MASK) { // Direct bit test
          next = D_IC_CLEAR_SUB(subst);
          continue;
        }
        break; // No substitution, so it's in WHNF
      }

      case CO0:
      case CO1: {
        // Collapser case - optimize for common path
        const uint32_t col_loc = D_IC_GET_VAL(next);
        const Term val = heap[col_loc];
        if (val & TERM_SUB_MASK) { // Direct bit test
          next = D_IC_CLEAR_SUB(val);
          continue;
        } else {
          // Direct push to stack
          stack[stack_pos++] = next;
          next = val;
          continue;
        }
      }

      case APP: {
        // Application case - optimize for this frequent operation
        const uint32_t app_loc = D_IC_GET_VAL(next);
        
        // Direct stack access
        stack[stack_pos++] = next;
        
        // Pre-load with software prefetch
        #if __CUDA_ARCH__ >= 700 
        // Use intrinsic prefetch for Volta+ architecture
        asm("prefetch.global.L1 [%0];" : : "l"(heap + app_loc));
        #else
        // Software prefetch for older architectures
        volatile Term temp = heap[app_loc];
        #endif
        
        next = heap[app_loc]; // Reduce the function part
        continue;
      }

      default: { // SUP, LAM
        // Handle default case (SUP, LAM) - optimize stack checks
        if (stack_pos == stop) {
          d_stack_pos = stack_pos; // Update stack position before return
          return next; // Stack empty, term is in WHNF
        } else {
          // Direct stack access 
          Term prev = stack[--stack_pos];
          
          // Get tag with optimized macro
          const TermTag ptag = D_IC_GET_TAG(prev);
          
          // Optimize interaction detection using direct bit comparison (combine tags)
          const uint32_t interaction_type = ((ptag << 3) | tag);
          
          // Fast interaction path for APP+LAM (most common)
          if (interaction_type == INTERACTION_APP_LAM) {
            next = d_ic_app_lam(prev, next);
            continue;
          } 
          // Fast interaction path for APP+SUP
          else if (interaction_type == INTERACTION_APP_SUP) {
            next = d_ic_app_sup(prev, next); 
            continue;
          }
          // CO0/CO1+LAM path
          else if (((ptag == CO0) || (ptag == CO1)) && tag == LAM) {
            next = d_ic_col_lam(prev, next);
            continue;
          }
          // CO0/CO1+SUP path
          else if (((ptag == CO0) || (ptag == CO1)) && tag == SUP) {
            next = d_ic_col_sup(prev, next);
            continue;
          }
          
          // No interaction found, return to stack
          stack[stack_pos++] = prev;
          break;
        }
      }
    }

    // After processing, check stack and update heap if needed
    if (stack_pos == stop) {
      d_stack_pos = stack_pos;
      return next; // Stack empty, return WHNF
    } else {
      // Process remaining stack
      while (stack_pos > stop) {
        // Direct stack access
        Term host = stack[--stack_pos];
        
        // Use optimized macros for faster extraction
        const TermTag htag = D_IC_GET_TAG(host);
        const uint32_t hloc = D_IC_GET_VAL(host);
        
        // Update the heap with the reduced term - only for specific tags
        if (htag == APP || htag == CO0 || htag == CO1) {
          heap[hloc] = next;
        }
        next = host;
      }
      d_stack_pos = stack_pos;
      return next; // Return updated original term
    }
  }
}

// Reduce a term to normal form - optimized version
__device__ __forceinline__ Term d_ic_normal(Term term) {
  // Reset stack
  d_stack_pos = 0;
  Term* const heap = d_heap;
  Term* const stack = d_stack;
  uint32_t stack_pos = 0;

  // No shared memory cache in this version

  // Allocate a new node for the initial term
  const uint32_t root_loc = d_ic_alloc(1);
  heap[root_loc] = term;

  // Push initial location to stack - use direct bit manipulation
  stack[stack_pos++] = (root_loc & TERM_VAL_MASK);

  // Main normalization loop - unroll initial iterations for better GPU performance
  #pragma unroll 8
  for (int i = 0; i < 8 && stack_pos > 0; i++) {
    // Pop current location from stack
    const uint32_t loc = stack[--stack_pos] & TERM_VAL_MASK;

    // Get term at this location with prefetch
    volatile Term temp = heap[loc];
    Term current = heap[loc];

    // Reduce to WHNF
    d_stack_pos = stack_pos;
    current = d_ic_whnf(current);
    stack_pos = d_stack_pos;

    // Store the WHNF term back to the heap
    heap[loc] = current;

    // Get term details - use direct bit manipulation
    const TermTag tag = (TermTag)((current & TERM_TAG_MASK) >> 28);
    const uint32_t val = current & TERM_VAL_MASK;

    // Push subterm locations based on term type
    if (tag == LAM) {
      stack[stack_pos++] = val & TERM_VAL_MASK;
    }
    else if (tag == APP || tag == SUP) {
      // Both APP and SUP need to push two locations
      stack[stack_pos++] = val & TERM_VAL_MASK;
      stack[stack_pos++] = (val + 1) & TERM_VAL_MASK;
    }
    // Other tags have no subterms to process
  }

  // Continue with remaining stack items
  while (stack_pos > 0) {
    // Pop current location from stack
    const uint32_t loc = stack[--stack_pos] & TERM_VAL_MASK;

    // Get term at this location
    Term current = heap[loc];

    // Reduce to WHNF
    d_stack_pos = stack_pos;
    current = d_ic_whnf(current);
    stack_pos = d_stack_pos;

    // Store the WHNF term back to the heap
    heap[loc] = current;

    // Get term details - use direct bit manipulation
    const TermTag tag = (TermTag)((current & TERM_TAG_MASK) >> 28);
    const uint32_t val = current & TERM_VAL_MASK;

    // Push subterm locations based on term type
    if (tag == LAM) {
      stack[stack_pos++] = val & TERM_VAL_MASK;
    }
    else if (tag == APP || tag == SUP) {
      // Both APP and SUP need to push two locations
      stack[stack_pos++] = val & TERM_VAL_MASK;
      stack[stack_pos++] = (val + 1) & TERM_VAL_MASK;
    }
    // Other tags have no subterms to process
  }

  // Update stack position and return the fully normalized term
  d_stack_pos = stack_pos;
  return heap[root_loc];
}

// CUDA kernel to normalize a term - optimized kernel configuration
__global__ void normalizeKernel() {
  // Single-threaded implementation (block 0, thread 0)
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    // No shared memory cache in this version
    
    // Get the term from the heap's entry point with prefetch hint
    volatile Term temp = d_heap[0];
    Term term = d_heap[0];
    
    // Perform normalization
    term = d_ic_normal(term);
    
    // Store the result back to the heap's entry point
    d_heap[0] = term;
  }
}

// Host function to normalize a term on the GPU - optimized memory transfers
extern "C" Term ic_normal_gpu(IC* ic, Term term) {
  // Debug outputs
  printf("GPU: Starting normalization\n");
  printf("GPU: Initial heap_pos = %u\n", ic->heap_pos);
  printf("GPU: Initial interactions = %llu\n", ic->interactions);
  
  // Allocate GPU memory for heap and stack with proper alignment for GPU
  Term* d_heap_ptr;
  Term* d_stack_ptr;
  uint32_t heap_size = ic->heap_size;
  uint32_t stack_size = ic->stack_size;
  
  // Use CUDA events to measure GPU time more accurately
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Select best GPU device if multiple are available
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount > 1) {
    int bestDevice = 0;
    int maxMultiprocessors = 0;
    hipDeviceProp_t prop;
    
    for (int device = 0; device < deviceCount; device++) {
      hipGetDeviceProperties(&prop, device);
      if (prop.multiProcessorCount > maxMultiprocessors) {
        maxMultiprocessors = prop.multiProcessorCount;
        bestDevice = device;
      }
    }
    
    hipSetDevice(bestDevice);
  }
  
  // Allocate device memory with proper alignment
  hipError_t err;
  printf("GPU: Allocating heap memory (%u terms, %zu bytes)\n", heap_size, heap_size * sizeof(Term));
  err = hipMalloc((void**)&d_heap_ptr, heap_size * sizeof(Term));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap allocation): %s\n", hipGetErrorString(err));
    return term; // Return original term on error
  }
  printf("GPU: Heap allocated successfully at %p\n", d_heap_ptr);
  
  err = hipMalloc((void**)&d_stack_ptr, stack_size * sizeof(Term));
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (stack allocation): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    return term; // Return original term on error
  }
  
  // Start timing
  hipEventRecord(start);
  
  // Copy only the needed portion of heap to device - use async copy for better performance
  hipStream_t stream;
  hipStreamCreate(&stream);
  err = hipMemcpyAsync(d_heap_ptr, ic->heap, ic->heap_pos * sizeof(Term), hipMemcpyHostToDevice, stream);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap copy to device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return term; // Return original term on error
  }
  
  // Set up constants on device
  uint32_t heap_pos = ic->heap_pos;
  uint64_t interactions = 0;
  uint32_t stack_pos = 0;
  
  // Use async copies with stream for better performance
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_heap), &d_heap_ptr, sizeof(Term*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_stack), &d_stack_ptr, sizeof(Term*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_heap_size), &heap_size, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_stack_size), &stack_size, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_heap_pos), &heap_pos, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_stack_pos), &stack_pos, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_interactions), &interactions, sizeof(uint64_t), 0, hipMemcpyHostToDevice, stream);
  
  // Make sure all async operations are complete before continuing
  hipStreamSynchronize(stream);
  
  // Configure kernel execution parameters
  // Use L1 cache preference for this memory-intensive application
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(normalizeKernel), hipFuncCachePreferL1);
  
  // Launch kernel with a single thread but optimal configuration
  // Use cuda occupancy API to get optimal block size
  int minGridSize;
  int blockSize;
  printf("GPU: Preparing kernel launch\n");
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, normalizeKernel, 0, 0);
  printf("GPU: Optimal block size: %d, grid size: %d\n", blockSize, minGridSize);
  
  // Launch with one thread but optimal SM configuration
  printf("GPU: Launching kernel\n");
  normalizeKernel<<<1, 1, 0, stream>>>();
  printf("GPU: Kernel launched\n");
  
  // Wait for kernel to complete
  printf("GPU: Waiting for kernel to complete\n");
  hipDeviceSynchronize();
  printf("GPU: Kernel execution completed\n");
  
  // Check for kernel errors
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Kernel Error: %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return term; // Return original term on error
  }
  printf("GPU: No kernel errors detected\n");
  
  // Get updated values back from device - use async for better performance
  hipMemcpyFromSymbolAsync(&heap_pos, HIP_SYMBOL(d_heap_pos), sizeof(uint32_t), 0, hipMemcpyDeviceToHost, stream);
  hipMemcpyFromSymbolAsync(&interactions, HIP_SYMBOL(d_interactions), sizeof(uint64_t), 0, hipMemcpyDeviceToHost, stream);
  
  // Wait for values to be available
  hipStreamSynchronize(stream);
  
  // Copy back only the used portion of the heap asynchronously
  err = hipMemcpyAsync(ic->heap, d_heap_ptr, heap_pos * sizeof(Term), hipMemcpyDeviceToHost, stream);
  
  // Wait for copy to complete
  hipStreamSynchronize(stream);
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error (heap copy from device): %s\n", hipGetErrorString(err));
    hipFree(d_heap_ptr);
    hipFree(d_stack_ptr);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return term; // Return original term on error
  }
  
  // Record stop time
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  
  // Update the host context
  ic->heap_pos = heap_pos;
  ic->interactions = interactions;
  
  // Free device memory and clean up resources
  hipFree(d_heap_ptr);
  hipFree(d_stack_ptr);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);
  
  // Return the normalized term
  return ic->heap[0];
}